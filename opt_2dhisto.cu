#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "opt_2dhisto.h"

// forward declarations
__global__ void opt_2dhisto_kernel(uint32_t *d_data, uint32_t *d_bins);
// end forward declarations

void opt_2dhisto(uint32_t *d_data, uint32_t *d_bins)
{
        static int gridsz = PADDED_INPUT_SIZE / BLOCK_SIZE + 1;
        static dim3 dimgrid(gridsz);
        static dim3 dimblock(BLOCK_SIZE);

        hipMemset(d_bins, 0, NUM_BINS * sizeof(uint32_t));

        opt_2dhisto_kernel<<<dimgrid, dimblock>>>(d_data, d_bins);
}

/* Include below the implementation of any other functions you need */

__global__ void opt_2dhisto_kernel(uint32_t *d_data, uint32_t *d_bins){
        const int globalTid = blockIdx.x * blockDim.x + threadIdx.x;

        const int numThreads = blockDim.x * gridDim.x;

        __shared__ data_tile[32];

        __shared__ local_hist[BLOCK_SIZE];

        // block local loading of shmem

        if(globalTid % PADDED_INPUT_WIDTH < INPUT_WIDTH){
                data_tile[threadIdx.x] = d_data[gobalTid];
        }

        // build local histogram out of shmem
        for(i = 0; i < 32; i++){
                local_hist[data_tile[(threadIdx.x + i) % 32]] += 1; // IMPLICITLY ASSUMES 1 WARP PER BLOCK
        }

        for(int pos = threadIdx.x; pos < NUM_BINS; pos += blockDim.x){
                d_bins[pos] = local_hist[pos];
        }
}

void setup(uint32_t **d_result, uint32_t **d_data, uint32_t **h_data)
{
        int grid_size = (NUM_BINS / BLOCK_SIZE) + 1;
        dim3 dimgrid(grid_size);
        dim3 dimblock(BLOCK_SIZE);

        hipMalloc((void **) d_result, NUM_BINS * sizeof(uint32_t));
        hipMalloc((void **) d_data, PADDED_INPUT_SIZE * sizeof(uint32_t));
        // pointers get mutated. Woo double indirection
        for(int i = 0; i < INPUT_HEIGHT; i++){
                hipMemcpy(*d_data + i * PADDED_INPUT_WIDTH,
                           (*h_data + i * PADDED_INPUT_WIDTH), // ignoring shitty outer array
                           PADDED_INPUT_WIDTH * sizeof(uint32_t),
                           hipMemcpyHostToDevice);
        }
}

void teardown(uint32_t *d_result, uint8_t *kernel_bins,  uint32_t *d_data)
{
        uint32_t *h_result = (uint32_t *) malloc(NUM_BINS * sizeof(uint32_t));
        hipMemcpy((void*) h_result,
                   (void*) d_result,
                   NUM_BINS * sizeof(uint32_t),
                   hipMemcpyDeviceToHost);

        for(int i = 0; i < NUM_BINS; i++){
                kernel_bins[i] = (h_result[i]); //> UINT8_MAX) ? 255 : h_result[i];
        }

        hipFree(d_data);
        hipFree(d_result);
}
